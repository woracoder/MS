#include <iostream>
#include <vector>
#include <stdio.h>
#include <algorithm>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <float.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sortBuckets(float *bucketsAsArrayD, int *eachBucketSizeArrayD) {

	//Copy to shared memory after initializing shared memory
	__shared__ float numsInBucket[4000];
	for(int i=0; i<4; i++) {
		if(i%1000 == threadIdx.x) {
			numsInBucket[i] = 10000000000.0;
		}
	}
	__syncthreads();

	int bucketSize = eachBucketSizeArrayD[blockIdx.x+1] - eachBucketSizeArrayD[blockIdx.x];
	int blockStart = eachBucketSizeArrayD[blockIdx.x];
	for(int i=0, j=0; i<4; i++) {
		if((j=(threadIdx.x + (i*1000))) < bucketSize) {
			numsInBucket[j] = bucketsAsArrayD[blockStart + j];
		}
	}
	__syncthreads();

	//Sort the bucket
	if(threadIdx.x == 0) {
		float key = 0.0;
		for(int j=1, i=0; j<bucketSize; j++) {
			key = numsInBucket[j];
			i = j - 1;
			while(i >= 0 and numsInBucket[i] > key) {
				numsInBucket[i + 1] = numsInBucket[i];
				i--;
			}
			numsInBucket[i + 1] = key;
		}
	}
	__syncthreads();

	//Copy back to global memory
	for(int i=0, j=0; i<4; i++) {
		if( (j=(threadIdx.x + (i*1000))) < bucketSize) {
			bucketsAsArrayD[blockStart + j] = numsInBucket[j];
		}
	}
}

vector< vector<float> > putNumbersIntoBuckets(vector< vector<float> > buckets, float nos[], int size, int argc);
double calculateMean(float nos[], int size);
double calculateStandardDeviation(float nos[], int size, float mean);
double cdf(double x);
int printSortedNumbers(float *nums, double time_spent, int size, int argc);

void random_number_generator_normal(float* arr, int size, int max_number);
void r4_nor_setup(uint32_t kn[128], float fn[128], float wn[128]);
float r4_nor(uint32_t *jsr, uint32_t kn[128], float fn[128], float wn[128]);
uint32_t shr3_seeded(uint32_t *jsr);
float r4_uni(uint32_t *jsr);

int main(int argc, char*argv[]) {

	if (argc < 3 || argc > 4) {
		cout << "Kindly enter -t, size of input and optional parameter (give 1) to print unsorted numbers, created bucket and sorted numbers in files." << endl;
		return EXIT_FAILURE;
	}

	struct timeval t;
	double start_t, end_t, time_spent;

	int size = atoi(argv[2]);
	float* nos = (float*) malloc(size * sizeof(float));
	random_number_generator_normal(nos, size, size);

	if(argc == 4) {
		FILE *un = fopen("UnsortedNosGpu.txt", "w");
		for(int i=0; i<size; i++) {
			fprintf(un, "%f\n", nos[i]);
		}
		fclose(un);
	}

	int noOfBuckets = (size/3000) + 1;
	vector< vector<float> > buckets(noOfBuckets, vector<float>(1));
	buckets = putNumbersIntoBuckets(buckets, nos, size, argc);
	free(nos);

	//Create a contiguous number list from buckets to copy to global memory
	float *bucketsAsArray = (float*)malloc(size * sizeof(float));
	int ptr = 0;
	for(int i=0, j=buckets.size(); i<j; i++) {
		for(int k=1, l=buckets.at(i).size(); k<l; k++) {
			bucketsAsArray[ptr++] = buckets.at(i).at(k);
		}
	}

	//Store size of each bucket in an array cumulatively
	int *eachBucketSizeArray = (int*) malloc((buckets.size()+1) * sizeof(int));
	eachBucketSizeArray[0] = 0;
	for(int i=0, j=buckets.size(); i<j; i++) {
		eachBucketSizeArray[i+1] = eachBucketSizeArray[i] + buckets.at(i).size() - 1;
	}

	//Make the call to kernel
	float *bucketsAsArrayD;
	int *eachBucketSizeArrayD;
	hipMalloc(&bucketsAsArrayD, size * sizeof(float));
	hipMalloc(&eachBucketSizeArrayD, (buckets.size()+1) * sizeof(int));
	hipMemcpy(bucketsAsArrayD, bucketsAsArray, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(eachBucketSizeArrayD, eachBucketSizeArray, (buckets.size()+1) * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(noOfBuckets);
	dim3 dimBlock(1000);

	gettimeofday(&t, NULL);
	start_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	sortBuckets<<<dimGrid, dimBlock>>> (bucketsAsArrayD, eachBucketSizeArrayD);

	hipMemcpy(bucketsAsArray, bucketsAsArrayD, size * sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&t, NULL);
	end_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	time_spent = end_t - start_t;

	hipFree(bucketsAsArrayD);
	hipFree(eachBucketSizeArrayD);

	printSortedNumbers(bucketsAsArray, time_spent, size, argc);
	return EXIT_SUCCESS;
}

vector< vector<float> > putNumbersIntoBuckets(vector< vector<float> > buckets, float nos[], int size, int argc) {
	double mean = calculateMean(nos, size);
	double stdDev = calculateStandardDeviation(nos, size, mean);
	double distanceFromStdDev = 0.0;
	for (int i = 0; i < size; i++) {
		distanceFromStdDev = (nos[i] - mean) / stdDev;
		double cdfVal = cdf(distanceFromStdDev);
		int bucketPosition = (int)(cdfVal * buckets.size());
		buckets.at(bucketPosition).push_back(nos[i]);
	}
	if(argc == 4) {
		FILE *ub = fopen("UnsortedBucketsGpu.txt", "w");
		for(int i=0; i<buckets.size(); i++) {
			fprintf(ub, "\nBucket %d size = %d\n\n", (i+1), buckets.at(i).size()-1);
		}
		fprintf(ub, "Bucket sizes list ends.\n\n");
		for(int i=0; i<buckets.size(); i++) {
			for(int j=1, k=buckets.at(i).size(); j<k; j++) {
				fprintf(ub, "%f\n", buckets.at(i).at(j));
			}
			fprintf(ub, "Bucket %d ends here with size = %d.\n\n", i+1, buckets.at(i).size()-1);
		}
		fclose(ub);
	}
	return buckets;
}

int printSortedNumbers(float* nums, double time_spent, int size, int argc) {
	FILE *tt = fopen("TimeTakenBucketSortGpu.txt", "a+");
	if (tt == NULL) {
		fprintf(stderr, "Can't open file to write output\n");
		return EXIT_FAILURE;
	}
	if(argc == 4) {
		FILE *sn = fopen("SortedNosGpu.txt", "w");
		if (sn == NULL) {
			fprintf(stderr, "Can't open file to write output\n");
			return EXIT_FAILURE;
		}
		for(int i=0; i<size; i++) {
			fprintf(sn, "%f\n", nums[i]);
		}
		fclose(sn);
	}
	fprintf(tt, "Total time required in seconds = %f for sorting %d numbers using GPU.\n", (time_spent / 1000000.0), size);
	fclose(tt);
	return EXIT_SUCCESS;
}

double calculateMean(float nos[], int size) {
	double sum = 0;
	for (int i = 0; i < size; i++) {
		sum += nos[i];
	}
	return (sum / size);
}

double calculateStandardDeviation(float nos[], int size, float mean) {
	double var = 0;
	for (int i = 0; i < size; i++) {
		var += pow((nos[i] - mean), 2.0);
	}
	var /= size;
	return (sqrt(var));
}

//http://www.johndcook.com/cpp_phi.html
double cdf(double x) {

    double a1 =  0.254829592;
    double a2 = -0.284496736;
    double a3 =  1.421413741;
    double a4 = -1.453152027;
    double a5 =  1.061405429;
    double p  =  0.3275911;

    int sign = 1;
    if (x < 0) {
        sign = -1;
    }
    x = fabs(x)/sqrt(2.0);

    double t = 1.0/(1.0 + p*x);
    double y = 1.0 - (((((a5*t + a4)*t) + a3)*t + a2)*t + a1)*t*exp(-x*x);

    return 0.5*(1.0 + sign*y);
}

void random_number_generator_normal(float* arr, int size, int max_number) {
	uint32_t kn[128];
	float fn[128], wn[128];
	r4_nor_setup(kn, fn, wn);
	float rnd;
	uint32_t seed = (uint32_t) time(NULL);
	float var = sqrt(max_number);
	for (int i = 0; i < size; i++) {
		rnd = r4_nor(&seed, kn, fn, wn);
		arr[i] = max_number / 2 + rnd * var;
	}
}

void r4_nor_setup(uint32_t kn[128], float fn[128], float wn[128]) {
	double dn = 3.442619855899;
	int i;
	const double m1 = 2147483648.0;
	double q;
	double tn = 3.442619855899;
	const double vn = 9.91256303526217E-03;

	q = vn / exp(-0.5 * dn * dn);

	kn[0] = (uint32_t) ((dn / q) * m1);
	kn[1] = 0;

	wn[0] = (float) (q / m1);
	wn[127] = (float) (dn / m1);

	fn[0] = 1.0;
	fn[127] = (float) (exp(-0.5 * dn * dn));

	for (i = 126; 1 <= i; i--) {
		dn = sqrt(-2.0 * log(vn / dn + exp(-0.5 * dn * dn)));
		kn[i + 1] = (uint32_t) ((dn / tn) * m1);
		tn = dn;
		fn[i] = (float) (exp(-0.5 * dn * dn));
		wn[i] = (float) (dn / m1);
	}

	return;
}

float r4_nor(uint32_t *jsr, uint32_t kn[128], float fn[128], float wn[128]) {
	int hz;
	uint32_t iz;
	const float r = 3.442620;
	float value;
	float x;
	float y;

	hz = (int) shr3_seeded(jsr);
	iz = (hz & 127);

	if (fabs(hz) < kn[iz]) {
		value = (float) (hz) * wn[iz];
	} else {
		for (;;) {
			if (iz == 0) {
				for (;;) {
					x = -0.2904764 * log(r4_uni(jsr));
					y = -log(r4_uni(jsr));
					if (x * x <= y + y) {
						break;
					}
				}

				if (hz <= 0) {
					value = -r - x;
				} else {
					value = +r + x;
				}
				break;
			}

			x = (float) (hz) * wn[iz];

			if (fn[iz] + r4_uni(jsr) * (fn[iz - 1] - fn[iz])
					< exp(-0.5 * x * x)) {
				value = x;
				break;
			}

			hz = (int) shr3_seeded(jsr);
			iz = (hz & 127);

			if (fabs(hz) < kn[iz]) {
				value = (float) (hz) * wn[iz];
				break;
			}
		}
	}

	return value;
}

uint32_t shr3_seeded(uint32_t *jsr) {
	uint32_t value;

	value = *jsr;

	*jsr = (*jsr ^ (*jsr << 13));
	*jsr = (*jsr ^ (*jsr >> 17));
	*jsr = (*jsr ^ (*jsr << 5));

	value = value + *jsr;

	return value;
}

float r4_uni(uint32_t *jsr) {
	uint32_t jsr_input;
	float value;

	jsr_input = *jsr;

	*jsr = (*jsr ^ (*jsr << 13));
	*jsr = (*jsr ^ (*jsr >> 17));
	*jsr = (*jsr ^ (*jsr << 5));

	value = fmod(0.5 + (float) (jsr_input + *jsr) / 65536.0 / 65536.0, 1.0);

	return value;
}
