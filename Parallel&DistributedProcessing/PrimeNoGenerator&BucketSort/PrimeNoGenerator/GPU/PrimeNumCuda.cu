#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

__global__ void generatePrimes(int *sizeD, int *maxPrimeD, int *numPrimesD) {
	__shared__ int numPrimesB;
	__shared__ int maxPrimeB;
	if(threadIdx.x == 0) {
		numPrimesB = 0;
		maxPrimeB = 0;
	}
	__syncthreads();
	int no = 9 + (blockIdx.x * 2000) + (2 * threadIdx.x);
	if(no < *sizeD) {
		int noRt = __double2int_ru(sqrt(__int2double_rn(no)));
		int k=3;
		for (k = 3; k <= noRt; k += 2) {
			if (no % k == 0) {
				break;
			}
		}
		if (k > noRt) {
			atomicAdd(&numPrimesB, 1);
			atomicMax(&maxPrimeB, no);
		}
		__syncthreads();
		if(threadIdx.x == 0) {
			atomicAdd(numPrimesD, numPrimesB);
			atomicMax(maxPrimeD, maxPrimeB);
		}
	}
}

int main(int argc, char* argv[]) {

	if (argc < 2 || argc > 3) {
		fprintf(stderr,
				"Please provide N: The problem size and/or optional flag(1) to print the prime numbers.\n");
	}

	struct timeval t;
	double start_t, end_t, time_spent;
	int i, maxPrime = 0, numPrimes = 0, size = atoi(argv[1]);
	int gridSize = (int)(floor(size/2000.0));
	dim3 dimGrid(gridSize+1);
	dim3 dimBlock(1000);

	int *sizeD, *maxPrimeD, *numPrimesD;
	hipMalloc((void**)&sizeD, sizeof(int));
	hipMalloc((void**)&maxPrimeD, sizeof(int));
	hipMalloc((void**)&numPrimesD, sizeof(int));

	gettimeofday(&t, NULL);
	start_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	hipMemcpy(sizeD, &size, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(maxPrimeD, &maxPrime, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(numPrimesD, &numPrimes, sizeof(int),hipMemcpyHostToDevice);
	generatePrimes<<<dimGrid, dimBlock>>> (sizeD, maxPrimeD, numPrimesD);
	hipMemcpy(&maxPrime, maxPrimeD, sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&numPrimes, numPrimesD, sizeof(int),hipMemcpyDeviceToHost);
	gettimeofday(&t, NULL);
	end_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	time_spent = end_t - start_t;

	hipFree(sizeD);
	hipFree(maxPrimeD);
	hipFree(numPrimesD);

	if(argc == 3) {
		FILE *pn;
		pn = fopen("PnGpuNos.txt", "w");
		if (pn == NULL) {
			fprintf(stderr, "Can't open file to write output\n");
			return EXIT_FAILURE;
		}
		fprintf(pn, "%d\n", 2);
		fprintf(pn, "%d\n", 3);
		fprintf(pn, "%d\n", 5);
		fprintf(pn, "%d\n", 7);
		for (i = 9; i < size; i += 2) {
			int sizeRt = ceil(sqrt(i));
			int k = 3;
			for (k = 3; k <= sizeRt; k += 2) {
				if (i % k == 0) {
					break;
				}
			}
			if (k > sizeRt) {
				maxPrime = i;
				fprintf(pn, "%d\n", i);
			}
		}
		fclose(pn);
	}

	FILE *tr = fopen("PnGpuTime.txt", "a+");
	if (tr == NULL) {
		fprintf(stderr, "Can't open file to write output\n");
		return EXIT_FAILURE;
	}
	fprintf(tr,
			"Maximum prime number = %d Total time taken using GPU for input size %d to generate %d prime numbers = %f secs\n",
			maxPrime, size, numPrimes+4, time_spent / 1000000.0);
	fclose(tr);

	return EXIT_SUCCESS;
}
